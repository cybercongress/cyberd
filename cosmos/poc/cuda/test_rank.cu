#include "rank.cu"
#include <stdint.h>



void test_getCompressedInLinksStartIndex() {

    uint32_t compressedInLinksCount [6] = { 0, 2, 0, 40, 13, 0 };
    uint64_t compressedInLinksStartIndex [6] = { };
    getCompressedInLinksStartIndex(6, compressedInLinksCount, compressedInLinksStartIndex);

    uint64_t expected [6] = {0,0,2,2,42,55};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(compressedInLinksStartIndex)))
        printf("getCompressedInLinksStartIndex() works as expected!\n");
    else {
        printf("getCompressedInLinksStartIndex() doesn't works :(\n");
        for (int i = sizeof(expected) / sizeof(expected[0])-1; i >= 0; i--)
            std::cout << compressedInLinksStartIndex[i] << ' ' << expected[i] << '\n';
    }
}

void test_getCompressedInLinksCount() {

    uint64_t cidsSize = 6;
    uint32_t inLinksCount [6] = { 0, 2, 0, 1, 3, 3 };
    uint64_t inLinksStartIndex [6] = { 0, 0, 2, 2, 3, 6 };
    uint64_t inLinksOuts [] = { 1, 1, 2, 2, 2, 1, 2, 1, 1};
    int outSize = (sizeof(inLinksOuts)/sizeof(*inLinksOuts));

    uint32_t *dev_inLinksCount;
    uint32_t *dev_compressedInLinksCount;
    uint64_t *dev_inLinksStartIndex;
    uint64_t *dev_inLinksOuts;

    hipMallocManaged(&dev_inLinksCount, cidsSize*sizeof(uint32_t));
    hipMallocManaged(&dev_compressedInLinksCount, cidsSize*sizeof(uint32_t));
    hipMallocManaged(&dev_inLinksStartIndex, cidsSize*sizeof(uint64_t));
    hipMallocManaged(&dev_inLinksOuts, outSize*sizeof(uint64_t));

    hipMemcpy(dev_inLinksCount, inLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksStartIndex, inLinksStartIndex, cidsSize*sizeof(uint64_t), hipMemcpyHostToDevice);
    hipMemcpy(dev_inLinksOuts, inLinksOuts, outSize*sizeof(uint64_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    getCompressedInLinksCount<<<1,6>>>(
        cidsSize,
        dev_inLinksStartIndex, dev_inLinksCount,
        dev_inLinksOuts, dev_compressedInLinksCount
    );
    hipDeviceSynchronize();

    uint32_t actual[6] = {};
    hipMemcpy(actual, dev_compressedInLinksCount, cidsSize*sizeof(uint32_t), hipMemcpyDeviceToHost);

    uint64_t expected[6] = {0,1,0,1,2,2};
    if (std::equal(std::begin(expected), std::end(expected), std::begin(actual)))
        printf("getCompressedInLinksCount() works as expected!\n");
    else {
       printf("getCompressedInLinksCount() doesn't works :(\n");
       for (int i = sizeof(actual) / sizeof(actual[0])-1; i >= 0; i--)
           std::cout << actual[i] << ' ' << expected[i] << '\n';
    }
}

// To run use `nvcc test_rank.cu -o test && ./test` command.
int main(void) {
    printf("Start testing !!!!!!!!!!!!!!!!!!\n");
    test_getCompressedInLinksStartIndex();
    test_getCompressedInLinksCount();
}
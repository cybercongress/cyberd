#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "types.h"

const double DUMP_FACTOR = 0.85;
const double TOLERANCE = 1e-3;

/*******************************************/
/* REPRESENTS INCOMING LINK WITH IT WEIGHT */
/*******************************************/
typedef struct {
    /* Index of opposite cid in cids array */
    uint64_t fromIndex;
    /* Index of user stake in stakes array */
    double weight;
} InLink;


/*****************************************************/
/* KERNEL: RUN SINGLE RANK ITERATION                 */
/*****************************************************/
/* For all given arrays, array index = cidId         */
/* Except: *inLinks, that represent 1D array of all  */
/*   links with corresponding weights                */
/*****************************************************/
__global__
void run_rank_iteration(
    InLink *inLinks,
    double *prevRank,
    double *rank,
    uint64_t *inLinksStartIndex,
    uint32_t *inLinksCount,
    uint64_t rankSize,
    double innerProductOverSize,
    double defaultRank
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < rankSize; i += stride) {
        double ksum = innerProductOverSize;
        for (uint64_t j = 0; j < inLinksCount[i]; j++) {
           // ksum = prevRank[inLinks[j].fromIndex] * inLinks[j].weight + ksum
           ksum = __fmaf_rz(prevRank[inLinks[j].fromIndex], inLinks[j].weight, ksum);
        }
        // rank[i] = ksum * DUMP_FACTOR + defaultRank
        rank[i] = __fmaf_rz(ksum, DUMP_FACTOR, defaultRank); // ksum * DUMP_FACTOR + defaultRank
    }
}


/*****************************************************/
/* KERNEL: DOUBLE ABS FUNCTOR                        */
/*****************************************************/
/* Return absolute value for double                  */
/*****************************************************/
struct absolute_value {
  __device__ double operator()(const double &x) const {
    return x < 0.0 ? -x : x;
  }
};


/*****************************************************/
/* KERNEL: FINDS MAXIMUM RANKS DIFFERENCE            */
/*****************************************************/
/* Finds maximum rank difference for single element  */
/*                                                   */
/*****************************************************/
double find_max_ranks_diff(double *prevRank, double *newRank, uint64_t rankSize) {

    thrust::device_vector<double> ranksDiff(rankSize);
    thrust::device_ptr<double> newRankBegin(newRank);
    thrust::device_ptr<double> prevRankBegin(prevRank);
    thrust::device_ptr<double> prevRankEnd(prevRank + rankSize);
    thrust::transform(thrust::device,
        prevRankBegin, prevRankEnd, newRankBegin, ranksDiff.begin(), thrust::minus<double>()
    );

    return thrust::transform_reduce(thrust::device,
        ranksDiff.begin(), ranksDiff.end(), absolute_value(), 0.0, thrust::maximum<double>()
    );
}

extern "C" {

    void calculate_rank(
        uint64_t *stakes, uint64_t stakesSize, /* User stakes and corresponding array size */
        cid *cids, uint64_t cidsSize, /* Cids links */
        cid_link *inLinks, cid_link *outLinks /* Incoming and Outgoing cids links */
    ) {

        printf("Cuda !!!!!!!!!!!!!!!!!!\n");

        double *prevRank, *rank;
        hipMalloc(&rank, cidsSize*sizeof(double));
        hipMalloc(&prevRank, cidsSize*sizeof(double));

        int steps = 0;
        double change = TOLERANCE + 1;
        while(change > TOLERANCE) {
        	//run_rank_iteration()
        	//change = calculateChange(prevrank, rank)
        	//prevrank = rank
        	steps++;
        	return;
        }

        hipFree(rank);
        hipFree(prevRank);
    }
};

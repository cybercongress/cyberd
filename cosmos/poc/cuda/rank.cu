#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include <thrust/transform.h>
#include <thrust/transform_reduce.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include "types.h"

const double DUMP_FACTOR = 0.85;
const double TOLERANCE = 1e-3;

/*****************************************************/
/* KERNEL: RUN SINGLE RANK ITERATION                 */
/*****************************************************/
/* All in links used here are compressed in links    */
/*****************************************************/
__global__
void run_rank_iteration(
    CompressedInLink *inLinks,                            /* all compressed in links */
    double *prevRank, double *rank,                       /* array index - cid index */
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,  /* array index - cid index */
    uint64_t rankSize,
    double innerProductOverSize, double defaultRank
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < rankSize; i += stride) {
        double ksum = innerProductOverSize;
        for (uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i] + inLinksCount[i]; j++) {
           // ksum = prevRank[inLinks[j].fromIndex] * inLinks[j].weight + ksum
           ksum = __fmaf_rz(prevRank[inLinks[j].fromIndex], inLinks[j].weight, ksum);
        }
        // rank[i] = ksum * DUMP_FACTOR + defaultRank
        rank[i] = __fmaf_rz(ksum, DUMP_FACTOR, defaultRank); // ksum * DUMP_FACTOR + defaultRank
    }
}


/*****************************************************/
/* KERNEL: DOUBLE ABS FUNCTOR                        */
/*****************************************************/
/* Return absolute value for double                  */
/*****************************************************/
struct absolute_value {
  __device__ double operator()(const double &x) const {
    return x < 0.0 ? -x : x;
  }
};


/*****************************************************/
/* HOST: FINDS MAXIMUM RANKS DIFFERENCE              */
/*****************************************************/
/* Finds maximum rank difference for single element  */
/*                                                   */
/*****************************************************/
double find_max_ranks_diff(double *prevRank, double *newRank, uint64_t rankSize) {

    thrust::device_vector<double> ranksDiff(rankSize);
    thrust::device_ptr<double> newRankBegin(newRank);
    thrust::device_ptr<double> prevRankBegin(prevRank);
    thrust::device_ptr<double> prevRankEnd(prevRank + rankSize);
    thrust::transform(thrust::device,
        prevRankBegin, prevRankEnd, newRankBegin, ranksDiff.begin(), thrust::minus<double>()
    );

    return thrust::transform_reduce(thrust::device,
        ranksDiff.begin(), ranksDiff.end(), absolute_value(), 0.0, thrust::maximum<double>()
    );
}

/*****************************************************/
/* KERNEL: CALCULATE CID TOTAL OUTS STAKE            */
/*****************************************************/
__global__
void calculateCidTotalOutStake(
    uint64_t cidsSize,
    uint64_t *stakes,                                        /*array index - user index*/
    uint64_t *outLinksStartIndex, uint32_t *outLinksCount,   /*array index - cid index*/
    uint64_t *outLinksUsers,                                 /*all out links from all users*/
    /*returns*/ uint64_t *cidsTotalOutStakes                 /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {
        uint64_t totalOutStake = 0;
        for (uint64_t j = outLinksStartIndex[i]; j < outLinksStartIndex[i] + outLinksCount[i]; j++) {
           totalOutStake += stakes[outLinksUsers[j]];
        }
        cidsTotalOutStakes[i] = totalOutStake;
    }
}

/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS COUNT FOR CIDS  */
/*********************************************************/
__global__
void getCompressedInLinksCount(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount,                    /*array index - cid index*/
    uint64_t *inLinksOuts,                                                  /*all incoming links from all users*/
    /*returns*/ uint32_t *compressedInLinksCount                            /*array index - cid index*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        uint32_t compressedLinksCount = 0;
        for(uint64_t j = inLinksStartIndex[i]; j < inLinksStartIndex[i]+inLinksCount[i]; j++) {
            if(j == inLinksStartIndex[i] || inLinksOuts[j] != inLinksOuts[j-1]) {
                compressedLinksCount++;
            }
        }
        compressedInLinksCount[i] = compressedLinksCount;
    }
}

/*********************************************************/
/* DEVICE: USER TO DIVIDE TWO uint64                     */
/*********************************************************/
__device__
double ddiv_rz(uint64_t *a, uint64_t *b) {
    return __ddiv_rz(__ull2double_rz(*a), __ull2double_rz(*b));
}


/*********************************************************/
/* KERNEL: CALCULATE COMPRESSED IN LINKS                 */
/*********************************************************/
__global__
void getCompressedInLinks(
    uint64_t cidsSize,
    uint64_t *inLinksStartIndex, uint32_t *inLinksCount, uint64_t *cidsTotalOutStakes,   /*array index - cid index*/
    uint64_t *inLinksOuts, uint64_t *inLinksUsers,                                       /*all incoming links from all users*/
    uint64_t *stakes,                                                                    /*array index - user index*/
    uint64_t *compressedInLinksStartIndex, uint32_t *compressedInLinksCount,             /*array index - cid index*/
    /*returns*/ CompressedInLink *compressedInLinks                                      /*all incoming compressed links*/
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    uint64_t stride = blockDim.x * gridDim.x;

    for (uint64_t i = index; i < cidsSize; i += stride) {

        if(inLinksCount[i] == 0) {
            continue;
        }

        uint32_t compressedLinksIndex = compressedInLinksStartIndex[i];

        if(inLinksCount[i] == 1) {
            uint64_t oppositeCid = inLinksOuts[inLinksStartIndex[i]];
            uint64_t compressedLinkStake = stakes[inLinksUsers[inLinksStartIndex[i]]];
            double weight = ddiv_rz(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
            compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
            continue;
        }

        uint64_t compressedLinkStake = 0;
        uint64_t lastLinkIndex = inLinksStartIndex[i] + inLinksCount[i] - 1;
        for(uint64_t j = inLinksStartIndex[i]; j < lastLinkIndex + 1; j++) {

            compressedLinkStake += stakes[inLinksUsers[j]];
            if(j == lastLinkIndex || inLinksOuts[j] != inLinksOuts[j+1]) {
                uint64_t oppositeCid = inLinksOuts[j];
                double weight = ddiv_rz(&compressedLinkStake, &cidsTotalOutStakes[oppositeCid]);
                compressedInLinks[compressedLinksIndex] = CompressedInLink {oppositeCid, weight};
                compressedLinksIndex++;
                compressedLinkStake=0;
            }
        }
    }
}

/************************************************************/
/* HOST: CALCULATE COMPRESSED IN LINKS START INDEXES        */
/************************************************************/
/* SEQUENTIAL LOGIC -> CALCULATE ON CPU                     */
/************************************************************/
__host__
void getCompressedInLinksStartIndex(
    uint64_t cidsSize,
    uint32_t *compressedInLinksCount,                   /*array index - cid index*/
    /*returns*/ uint64_t *compressedInLinksStartIndex   /*array index - cid index*/
) {

    uint64_t index = 0;
    for (uint64_t i = 0; i < cidsSize; i++) {
        compressedInLinksStartIndex[i] = index;
        index += compressedInLinksCount[i];
    }
}

extern "C" {

    void calculate_rank(
        uint64_t *stakes, uint64_t stakesSize, /* User stakes and corresponding array size */
        cid *cids, uint64_t cidsSize, /* Cids links */
        cid_link *inLinks, cid_link *outLinks /* Incoming and Outgoing cids links */
    ) {

        /*-------------------------------------------------------------------*/
        printf("Cuda !!!!!!!!!!!!!!!!!!\n");
        printf("Initializing device memory\n");

        uint64_t *cidsTotalOutStakes; // for each cid sum of all out links stake
        uint32_t *compressedInLinksCount; // for each cid count of compressed links

        hipMalloc(&cidsTotalOutStakes, cidsSize*sizeof(uint64_t));
        hipMalloc(&compressedInLinksCount, cidsSize*sizeof(uint32_t));
        //todo

        hipFree(cidsTotalOutStakes);
        hipFree(compressedInLinksCount);

        /*-------------------------------------------------------------------*/
        printf("Calculating rank\n");

        double *prevRank, *rank;
        hipMalloc(&rank, cidsSize*sizeof(double));
        hipMalloc(&prevRank, cidsSize*sizeof(double));

        int steps = 0;
        double change = TOLERANCE + 1;
        while(change > TOLERANCE) {
        	//run_rank_iteration()
        	//change = find_max_ranks_diff(prevrank, rank, cidsSize);
        	//prevrank = rank
        	steps++;
        	return;
        }

        hipFree(rank);
        hipFree(prevRank);
    }
};

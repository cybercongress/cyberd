#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdio.h>
#include "types.h"


/******************************************/
/* CELL STRUCT LEADING TO ARRAY OF STRUCT */
/******************************************/
typedef struct {
    /* Index of opposite cid in cids array */
    uint64_t fromIndex;
    /* Index of user stake in stakes array */
    uint32_t weight;
} InLink;


/*****************************************************/
/* KERNEL: RUN SINGLE RANK ITERATION                         */
/*****************************************************/
/* For all given arrays, array index = cidId         */
/* Except: *inLinks, that represent 1d array of all  */
/*   i->j links with corresponding weights           */
/*****************************************************/
__global__
void run_rank_iteration(
    InLink *inLinks,
    uint64_t *prevRank,
    uint64_t *rank,
    uint64_t *inLinksStartIndex,
    uint32_t *inLinksCount,
    uint64_t rankSize,
    uint64_t innerProductOverSize,
    uint64_t defaultRank
) {

	int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < rankSize; i += stride) {
        rank[i] = innerProductOverSize;
        for (int j = 0; j < inLinksCount[i]; j++) {
           rank[i] += prevRank[inLinks[j].fromIndex] * inLinks[j].weight;
        }
        rank[i] = rank[i] / 20 * 17 + defaultRank;
    }
}

/******************************************/
/* CELL STRUCT LEADING TO ARRAY OF STRUCT */
/******************************************/
extern "C" {

    void calculate_rank(
        uint64_t *stakes, uint64_t stakesSize, /* User stakes and corresponding array size */
        cid *cids, uint64_t cidsSize, /* Cids links */
        cid_link *inLinks, cid_link *outLinks /* Incoming and Outgoing cids links */
    ) {


    }
};
